#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "cudaHelpers.cuh"

__global__ void sharedMemBandwidthKernel(uint32_t numElems, uint32_t numIters, uint32_t elemBytes) {
  extern __shared__ uint8_t sharedMemRaw[];
  uint32_t* sharedMem = reinterpret_cast<uint32_t*>(sharedMemRaw);
  const uint32_t tid = threadIdx.x;
  uint32_t tmp = 0u;

  __syncthreads();

  for (uint32_t i = 0; i < numIters; ++i) {
    /* random offset */
    const uint32_t offset = (tid * 37 + i * 17) % numElems;
    /* write */
    sharedMem[offset] = tid + i;
    /* read */
    tmp += sharedMem[offset];
  }

  /* Prevent compiler optimization. */
  if (tmp == 0xFFFFFFFFu)
    sharedMem[0] = tmp;
  __syncthreads();
}

void launchSharedMemBandwidthKernel(uint32_t numElems, uint32_t numIters, uint32_t threads, size_t sharedBytes,
                                    float* elapsedMsOut) {
  hipError_t err;

  hipEvent_t start, stop;
  err = hipEventCreate(&start);
  throwOnErr(err);
  err = hipEventCreate(&stop);
  throwOnErr(err);

  const dim3 grid(1);
  const dim3 block(threads);

  /* Record start event. */
  err = hipEventRecord(start);
  throwOnErr(err);

  // Kernel uses extern shared memory size = sharedBytes
  sharedMemBandwidthKernel<<<grid, block, sharedBytes>>>(numElems, numIters, 4);

  /* Record stop event. */
  err = hipEventRecord(stop);
  throwOnErr(err);

  err = hipEventSynchronize(stop);
  throwOnErr(err);

  float ms = 0.0f;
  err = hipEventElapsedTime(&ms, start, stop);
  throwOnErr(err);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  err = hipGetLastError();
  throwOnErr(err);

  if (elapsedMsOut)
    *elapsedMsOut = ms;
}
