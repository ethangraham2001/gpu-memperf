#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <nvml.h>

#include <cudaHelpers.cuh>
#include <stdexcept>

__global__ void clock64OverheadKernel(uint64_t iters, uint64_t* cycles) {
  uint64_t total = 0;

  /* Warmup - clock64() should be in instruction cache. For the benchmark */
  for (uint64_t i = 0; i < iters; i++) {
    uint64_t start = clock64();
    uint64_t end = clock64();
    total += (end - start);
  }
  /* Prevent optimization. */
  if (total == UINT64_MAX) {
    *cycles = total;
    return;
  }

  total = 0;
  for (uint64_t i = 0; i < iters; i++) {
    uint64_t start = clock64();
    uint64_t end = clock64();
    total += (end - start);
  }
  *cycles = total;
}

double measureClock64Latency(uint64_t iters) {
  hipError_t err;
  uint64_t* dCycles;
  uint64_t hCycles;

  err = hipMalloc(&dCycles, sizeof(uint64_t));
  throwOnErr(err);

  clock64OverheadKernel<<<1, 1>>>(iters, dCycles);

  err = hipDeviceSynchronize();
  throwOnErr(err);

  err = hipGetLastError();
  throwOnErr(err);

  err = hipMemcpy(&hCycles, dCycles, sizeof(uint64_t), hipMemcpyDeviceToHost);
  throwOnErr(err);

  return (double)hCycles / ((double)iters * 2.0);
}

unsigned int getMaxClockFrequencyHz() {
  nvmlInit();
  nvmlDevice_t dev;
  nvmlReturn_t ret = nvmlDeviceGetHandleByIndex(0, &dev);
  if (ret != NVML_SUCCESS)
    throw std::runtime_error("Unable to find device");

  unsigned int clockMHz;
  ret = nvmlDeviceGetMaxClockInfo(dev, NVML_CLOCK_SM, &clockMHz);
  if (ret != NVML_SUCCESS)
    throw std::runtime_error("Unable to read device's clock frequency");

  nvmlShutdown();
  return clockMHz * 1000000U;
}
