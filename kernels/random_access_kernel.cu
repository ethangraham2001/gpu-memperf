#include "hip/hip_runtime.h"
#include <vector>

#include <hip/hip_runtime.h>

#include <Types.hh>
#include <cudaHelpers.cuh>
#include <random_access_kernel.hh>

/**
 * l1LoadElem - inline PTX for loading an element from L1 cache
 *
 * Constexpr templating means that this if statement is never evaluated at
 * runtime on the device - it is evaluated statically.
 */
template <typename T>
__device__ __forceinline__ void l1LoadElem(T* addr, uint64_t& sink) {
  if constexpr (sizeof(T) == sizeof(types::f8)) {
    asm volatile("{\t\n .reg .u64 data64;\n\t ld.global.ca.u8 data64, [%1];\n\t add.u64 %0, %0, data64;\n\t }"
                 : "+l"(sink)
                 : "l"(addr)
                 : "memory");
  } else if constexpr (sizeof(T) == sizeof(types::f16)) {
    asm volatile("{\t\n .reg .u64 data64;\n\t ld.global.ca.u16 data64, [%1];\n\t add.u64 %0, %0, data64;\n\t }"
                 : "+l"(sink)
                 : "l"(addr)
                 : "memory");
  } else if constexpr (sizeof(T) == sizeof(types::f32)) {
    asm volatile("{\t\n .reg .u64 data64;\n\t ld.global.ca.u32 data64, [%1];\n\t add.u64 %0, %0, data64;\n\t }"
                 : "+l"(sink)
                 : "l"(addr)
                 : "memory");
  } else if constexpr (sizeof(T) == sizeof(types::f64)) {
    asm volatile("{\t\n .reg .u64 data64;\n\t ld.global.ca.u64 data64, [%1];\n\t add.u64 %0, %0, data64;\n\t }"
                 : "+l"(sink)
                 : "l"(addr)
                 : "memory");
  }
}

/**
 * randomAccessKernelL1 - L1 random access bandwidth measurement kernel
 *
 * The host must provide some set of random indices with the same cardinality as
 * the data array which should be a permutation of range [0, numElems - 1].
 *
 * Access to the indices array is perfectly coalesced for maximum efficiency,
 * and the indices contained within it should be well-distributed so that
 * accesses are random.
 *
 * @data: a data array of type T, which whould be a primitive type
 * @indices: a random permutation of range [0, numElems - 1]
 * @numElems: cardinality of @data and @indices
 * @numAccesses: the number of accesses performed per thread
 * @results[numThreads]: number of cycles per thread for the accesses
 * @totalCycles: cycles for all threads to complete as measured by thread 0
 * @sink[numThreads]: used to prevent compiler optimization - can be ignored
 */
template <typename T>
__global__ void randomAccessKernelL1(T* data, uint32_t* indices, uint64_t numElems, uint64_t numAccesses,
                                     uint64_t* results, uint64_t* totalCycles, uint64_t* sink) {
  uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t totalThreads = gridDim.x * blockDim.x;
  /* Shared memory to ensure that exactly one copy of each of these exists. */
  __shared__ uint64_t sharedStart, sharedEnd;
  uint64_t localSink = 0;

  /* Warm the cache. */
  for (uint64_t i = 0; i < numAccesses; i++) {
    uint64_t idx = indices[(tid + i * totalThreads) mod_power_of_2(numElems)];
    l1LoadElem(&data[idx], localSink);
  }

  /* Sync on both ends of launching the timer for best accuracy - we don't want
   * any threads to begin accessing data until the timer has started. */
  __syncthreads();
  if (threadIdx.x == 0)
    sharedStart = clock64();
  __syncthreads();

  uint64_t start = clock64();
  for (uint64_t i = 0; i < numAccesses; i++) {
    uint64_t idx = indices[(tid + i * totalThreads) mod_power_of_2(numElems)];
    l1LoadElem(&data[idx], localSink);
  }
  uint64_t end = clock64();

  /* Sync before stopping the global timer. Similarly to above, we don't want to
   * be stopping the timer until all threads have finished their work. */
  __syncthreads();
  if (threadIdx.x == 0) {
    sharedEnd = clock64();
    *totalCycles = sharedEnd - sharedStart;
  }

  results[tid] = end - start;
  sink[tid] = localSink;
}

template <typename T>
__global__ void randomAccessKernelL2(T* data, uint32_t* indices, uint64_t numElems, uint64_t numAccesses,
                                     uint64_t* results, uint64_t* sharedCycles, uint64_t* sink) { /* TODO */ }

template <typename T>
__global__ void randomAccessKernelDRAM(T* data, uint32_t* indices, uint64_t numElems, uint64_t numAccesses,
                                       uint64_t* results, uint64_t* sharedCycles, uint64_t* sink) { /* TODO */ }

template <typename T>
using randomAccessKernelFunc = void (*)(T*, uint32_t*, uint64_t, uint64_t, uint64_t*, uint64_t*, uint64_t*);

template <typename T>
static randomAccessKernelFunc<T> getKernel(randomAccessKernel::mode mode) {
  switch (mode) {
    case randomAccessKernel::L1_CACHE:
      return randomAccessKernelL1<T>;
    case randomAccessKernel::L2_CACHE:
      return randomAccessKernelL2<T>;
    case randomAccessKernel::DRAM:
      return randomAccessKernelDRAM<T>;
    default:
      throw std::invalid_argument("invalid mode");
  }
}

template <typename T>
std::pair<uint64_t, std::vector<uint64_t>> launchRandomAccessKernel(const std::vector<T>& data,
                                                                    const std::vector<uint32_t>& indices,
                                                                    uint64_t numAccesses, uint64_t threadsPerBlock,
                                                                    uint64_t numBlocks, randomAccessKernel::mode mode) {
  uint64_t* dTimingResults;
  uint64_t* dSharedCycles;
  uint32_t* dIndices;
  uint64_t* dSink;
  T* dData;

  uint64_t totalThreads = numBlocks * threadsPerBlock;

  throwOnErr(hipMalloc(&dData, data.size() * sizeof(T)));
  throwOnErr(hipMemcpy(dData, data.data(), data.size() * sizeof(T), hipMemcpyHostToDevice));

  throwOnErr(hipMalloc(&dIndices, indices.size() * sizeof(uint32_t)));
  throwOnErr(hipMemcpy(dIndices, indices.data(), indices.size() * sizeof(uint32_t), hipMemcpyHostToDevice));

  throwOnErr(hipMalloc(&dSink, totalThreads * sizeof(uint64_t)));
  throwOnErr(hipMalloc(&dTimingResults, totalThreads * sizeof(uint64_t)));
  throwOnErr(hipMalloc(&dSharedCycles, sizeof(uint64_t)));

  auto kernel = getKernel<T>(mode);
  kernel<<<static_cast<unsigned int>(numBlocks), static_cast<unsigned int>(threadsPerBlock)>>>(
      dData, dIndices, data.size(), numAccesses, dTimingResults, dSharedCycles, dSink);

  throwOnErr(hipDeviceSynchronize());

  throwOnErr(hipGetLastError());

  uint64_t* hTimingResults = static_cast<uint64_t*>(malloc(threadsPerBlock * numBlocks * sizeof(uint64_t)));
  throwOnErr(hipMemcpy(hTimingResults, dTimingResults, totalThreads * sizeof(uint64_t), hipMemcpyDeviceToHost));

  uint64_t hSharedCycles;
  throwOnErr(hipMemcpy(&hSharedCycles, dSharedCycles, sizeof(uint64_t), hipMemcpyDeviceToHost));

  hipFree(dData);
  hipFree(dTimingResults);
  hipFree(dSink);

  std::vector<uint64_t> result(totalThreads);
  for (uint64_t i = 0; i < totalThreads; i++)
    result[i] = hTimingResults[i];
  return {hSharedCycles, result};
}

/* The compiler complains when the concrete versions that we use aren't defined.
 * The implementation isn't required - just a header. So we declare a concrete
 * header for each version of randomAccessKernel that we use. */
template std::pair<uint64_t, std::vector<uint64_t>> launchRandomAccessKernel<types::f8>(const std::vector<types::f8>&,
                                                                                        const std::vector<uint32_t>&,
                                                                                        uint64_t, uint64_t, uint64_t,
                                                                                        randomAccessKernel::mode);
template std::pair<uint64_t, std::vector<uint64_t>> launchRandomAccessKernel<types::f16>(const std::vector<types::f16>&,
                                                                                         const std::vector<uint32_t>&,
                                                                                         uint64_t, uint64_t, uint64_t,
                                                                                         randomAccessKernel::mode);
template std::pair<uint64_t, std::vector<uint64_t>> launchRandomAccessKernel<types::f32>(const std::vector<types::f32>&,
                                                                                         const std::vector<uint32_t>&,
                                                                                         uint64_t, uint64_t, uint64_t,
                                                                                         randomAccessKernel::mode);
template std::pair<uint64_t, std::vector<uint64_t>> launchRandomAccessKernel<types::f64>(const std::vector<types::f64>&,
                                                                                         const std::vector<uint32_t>&,
                                                                                         uint64_t, uint64_t, uint64_t,
                                                                                         randomAccessKernel::mode);
